#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define J 6
#define K 2.0
#define T_eq 1.0
#define a 0.01
#define alpha 0.9
#define delta 0.04
#define dt 0.0001
#define eps_bar 0.01
#define gamma 10.0
#define t_OFF 0.50
#define tau 0.0003
#define nIter int(t_OFF/dt)+1

const int nx = 300; 
const int ny = 300;
const float hx = 0.03f;
const float hy = 0.03f;

// Error checking macro
#define CUDA_CHECK_ERROR() {                                           \
    hipError_t err = hipGetLastError();                              \
    if (err != hipSuccess) {                                          \
        printf("CUDA Error: %s\n", hipGetErrorString(err));           \
        exit(-1);                                                      \
    }                                                                  \
}

__global__ void init_curand(hiprandState* state, unsigned long seed, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;
    if (idx < nx && idy < ny) {
        hiprand_init(seed, id, 0, &state[id]);
    }
}

__global__ void grad(float* m, float* f_x, float* f_y, float dx, float dy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;
        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;
        f_x[id] = (m[right] - m[left]) / (2.0 * dx);
        f_y[id] = (m[bottom] - m[top]) / (2.0 * dy);
    }
}

__global__ void laplace(float* m, float* result, float hx, float hy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;

        result[id] = (m[top] + m[bottom] - 2.0 * m[id]) / (hx * hx) + 
                     (m[left] + m[right] - 2.0 * m[id]) / (hy * hy);
    }
}

__global__ void get_theta(float* f_x, float* f_y, float* theta, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        theta[id] = 0.0;
        if (f_x[id] == 0 && f_y[id] > 0)
            theta[id] = 0.5 * M_PI;
        else if (f_x[id] == 0 && f_y[id] < 0)
            theta[id] = 1.5 * M_PI;
        else if (f_x[id] > 0 && f_y[id] < 0)
            theta[id] = 2 * M_PI + atan(f_y[id] / f_x[id]);
        else if (f_x[id] > 0 && f_y[id] > 0)
            theta[id] = atan(f_y[id] / f_x[id]);
        else if (f_x[id] < 0)
            theta[id] = M_PI + atan(f_y[id] / f_x[id]);
    }
}

__global__ void get_eps(float* theta, float* eps, float* eps_prime, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        eps[id] = eps_bar * (1 + delta * cos(J * theta[id]));
        eps_prime[id] = -eps_bar * J * delta * sin(J * theta[id]);
    }
}

__global__ void phase_field(hiprandState* state, float* eps, float* eps_prime, float* eps2_x, float* eps2_y, float* p, float* p_x, float* p_y, float* p_lap, float* p_dif, float* T, float dx, float dy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;
    if (idx < nx && idy < ny) {

        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;

        float part1 = (eps[right] * eps_prime[right] * p_y[right] - eps[left] * eps_prime[left] * p_y[left]) / (2.0 * dx);
        float part2 = (eps[bottom] * eps_prime[bottom] * p_x[bottom] - eps[top] * eps_prime[top] * p_x[top]) / (2.0 * dy);
        float part3 = eps2_x[id] * p_x[id] + eps2_y[id] * p_y[id];
        float part4 = eps[id] * eps[id] * p_lap[id];

        float m = alpha / M_PI * atan(gamma * (T_eq - T[id]));

        float term1 = -part1 + part2 + part3 + part4;
        float term2 = p[id] * (1 - p[id]) * (p[id] - 0.5 + m);

        // Add noise using curand
        hiprandState localState = state[id];
        float noise = a * p[id] * (1 - p[id]) * (hiprand_uniform(&localState) - 0.5);
        
        p_dif[id] = dt / tau * (term1 + term2 + noise);

        p[id] = p[id] + p_dif[id];
    }
}

__global__ void T_field(float* T, float* p_dif, float* T_lap, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;
        T[id] = T[id] + dt*T_lap[id] + K*p_dif[id];
    }
}

__global__ void zero_flux_BC(float* arr, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        if (idx == 0)
            arr[id] = arr[idy * nx + 1];
        else if (idx == nx - 1)
            arr[id] = arr[idy * nx + (nx - 2)];
        if (idy == 0)
            arr[id] = arr[1 * nx + idx];
        else if (idy == ny - 1)
            arr[id] = arr[(ny - 2) * nx + idx];
    }
}

__global__ void elementwise_multiply(float* aa, float* bb, float* result, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;

    if (idx < nx && idy < ny) {
        result[id] = aa[id] * bb[id];
    }
}

void printArray(const char* name, float* array, int nx, int ny) {
    std::cout << name << ":\n";
    for (int i = 0; i < ny; ++i) {
        for (int j = 0; j < nx; ++j) {
            std::cout << array[i * nx + j] << " ";
        }
        std::cout << "\n";
    }
}

void saveArray(const char* filename, float* array, int nx, int ny) {
    std::ofstream file(filename);
    for (int i = 0; i < ny; ++i) {
        for (int j = 0; j < nx; ++j) {
            file << array[i * nx + j] << " ";
        }
        file << "\n";
    }
    file.close();
}

int main() {
    float t = 0.0;

    // Initialize arrays
    float *T, *p, *theta, *p_x, *p_y, *eps, *eps_prime, *eps2_x, *eps2_y;
    float *d_T, *d_T_lap, *d_p, *d_theta, *d_p_x, *d_p_y, *d_p_dif, *d_p_lap, *d_eps, *d_eps2, *d_eps_prime, *d_eps2_x, *d_eps2_y;
    hiprandState* d_state;

    T = (float*)calloc(nx * ny, sizeof(float));
    p = (float*)calloc(nx * ny, sizeof(float));
    theta = (float*)calloc(nx * ny, sizeof(float));
    p_x = (float*)calloc(nx * ny, sizeof(float));
    p_y = (float*)calloc(nx * ny, sizeof(float));
    eps = (float*)calloc(nx * ny, sizeof(float));
    eps_prime = (float*)calloc(nx * ny, sizeof(float));
    eps2_x = (float*)calloc(nx * ny, sizeof(float));
    eps2_y = (float*)calloc(nx * ny, sizeof(float));

    // Define the center of the grid
    int centerX = nx / 2;
    int centerY = ny / 2;
    int radius = 5;

    // Loop over each element in the grid
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            // Check if the current element is within the nucleation region
            if ((i - centerX) * (i - centerX) + (j - centerY) * (j - centerY) < radius * radius) {
                // Set the value to 1 for nucleation
                p[j * nx + i] = 1.0;
            }
        }
    }

    hipMalloc((void**)&d_T, nx * ny * sizeof(float));
    hipMalloc((void**)&d_T_lap, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_dif, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_lap, nx * ny * sizeof(float));
    hipMalloc((void**)&d_theta, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_x, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_y, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps_prime, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2_x, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2_y, nx * ny * sizeof(float));
    hipMalloc((void**)&d_state, nx * ny * sizeof(hiprandState));

    hipMemcpy(d_T, T, nx * ny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, nx * ny * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    init_curand<<<numBlocks, threadsPerBlock>>>(d_state, time(0), nx, ny);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();

    for (int i = 0; i <= nIter; ++i) {
        grad<<<numBlocks, threadsPerBlock>>>(d_p, d_p_x, d_p_y, hx, hy, nx, ny);
        hipDeviceSynchronize();

        laplace<<<numBlocks, threadsPerBlock>>>(d_p, d_p_lap, hx, hy, nx, ny);
        hipDeviceSynchronize();

        get_theta<<<numBlocks, threadsPerBlock>>>(d_p_x, d_p_y, d_theta, nx, ny);
        hipDeviceSynchronize();

        get_eps<<<numBlocks, threadsPerBlock>>>(d_theta, d_eps, d_eps_prime, nx, ny);
        hipDeviceSynchronize();

        elementwise_multiply<<<numBlocks, threadsPerBlock>>>(d_eps, d_eps, d_eps2, nx, ny);
        hipDeviceSynchronize();

        grad<<<numBlocks, threadsPerBlock>>>(d_eps2, d_eps2_x, d_eps2_y, hx, hy, nx, ny);
        hipDeviceSynchronize();

        phase_field<<<numBlocks, threadsPerBlock>>>(d_state, d_eps, d_eps_prime, d_eps2_x, d_eps2_y, d_p, d_p_x, d_p_y, d_p_lap, d_p_dif, d_T, hx, hy, nx, ny);
        hipDeviceSynchronize();

        zero_flux_BC<<<numBlocks, threadsPerBlock>>>(d_p, nx, ny);
        hipDeviceSynchronize();

        laplace<<<numBlocks, threadsPerBlock>>>(d_T, d_T_lap, hx, hy, nx, ny);
        hipDeviceSynchronize();

        T_field<<<numBlocks, threadsPerBlock>>>(d_T, d_p_dif, d_T_lap, nx, ny);
        hipDeviceSynchronize();

        zero_flux_BC<<<numBlocks, threadsPerBlock>>>(d_T, nx, ny);
        hipDeviceSynchronize();

        if (i % 800 == 0) {
            std::cout << "step/"<<nIter<<": " << i << "\n";
            hipMemcpy(p, d_p, nx * ny * sizeof(float), hipMemcpyDeviceToHost);

            char filename[100];
            sprintf(filename, "ice/output_%d.txt", i);
            saveArray(filename, p, nx, ny);
        }

        t += dt;
    }

    hipFree(d_T);
    hipFree(d_p);
    hipFree(d_theta);
    hipFree(d_p_x);
    hipFree(d_p_y);
    hipFree(d_eps);
    hipFree(d_eps_prime);
    hipFree(d_eps2_x);
    hipFree(d_eps2_y);
    hipFree(d_state);

    free(T);
    free(p);
    free(theta);
    free(p_x);
    free(p_y);
    free(eps);
    free(eps_prime);
    free(eps2_x);
    free(eps2_y);

    std::cout << "t=" << t << std::endl;
    return 0;
}
