#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define J 6
#define K 1.6
#define T_eq 1.0
#define a 0.01
#define alpha 0.9
#define delta 0.04
#define dt 0.0001
#define eps_bar 0.01
#define gamma 10.0
#define t_OFF 0.36
#define tau 0.0003
#define nIter int(t_OFF/dt)

const int nx = 300; // Use const or constexpr to ensure they are properly handled
const int ny = 300;
const float hx = 0.03f;
const float hy = 0.03f;

// Error checking macro
#define CUDA_CHECK_ERROR() {                                           \
    hipError_t err = hipGetLastError();                              \
    if (err != hipSuccess) {                                          \
        printf("CUDA Error: %s\n", hipGetErrorString(err));           \
        exit(-1);                                                      \
    }                                                                  \
}

__global__ void init_curand(hiprandState* state, unsigned long seed, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;
    if (idx < nx && idy < ny) {
        hiprand_init(seed, id, 0, &state[id]);
    }
}

__global__ void grad(float* m, float* f_x, float* f_y, float dx, float dy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;
        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;
        f_x[id] = (m[right] - m[left]) / (2.0 * dx);
        f_y[id] = (m[bottom] - m[top]) / (2.0 * dy);
    }
}

__global__ void laplace(float* m, float* result, float hx, float hy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;

        result[id] = (m[top] + m[bottom] - 2.0 * m[id]) / (hx * hx) + 
                     (m[left] + m[right] - 2.0 * m[id]) / (hy * hy);
    }
}

__global__ void get_theta(float* f_x, float* f_y, float* theta, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        theta[id] = 0.0;
        if (f_x[id] == 0 && f_y[id] > 0)
            theta[id] = 0.5 * M_PI;
        else if (f_x[id] == 0 && f_y[id] < 0)
            theta[id] = 1.5 * M_PI;
        else if (f_x[id] > 0 && f_y[id] < 0)
            theta[id] = 2 * M_PI + atan(f_y[id] / f_x[id]);
        else if (f_x[id] > 0 && f_y[id] > 0)
            theta[id] = atan(f_y[id] / f_x[id]);
        else if (f_x[id] < 0)
            theta[id] = M_PI + atan(f_y[id] / f_x[id]);
    }
}

__global__ void get_eps(float* theta, float* eps, float* eps_prime, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        eps[id] = eps_bar * (1 + delta * cos(J * theta[id]));
        eps_prime[id] = -eps_bar * J * delta * sin(J * theta[id]);
    }
}

__global__ void phase_field(hiprandState* state, float* eps, float* eps_prime, float* eps2_x, float* eps2_y, float* p, float* p_x, float* p_y, float* p_lap, float* dif_p, float* T, float dx, float dy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;
    if (idx < nx && idy < ny) {

        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;

        float part1 = (eps[id] * eps_prime[id] * p_y[right] - eps[id] * eps_prime[id] * p_y[left]) / (2.0 * dx);
        float part2 = (eps[id] * eps_prime[id] * p_x[bottom] - eps[id] * eps_prime[id] * p_x[top]) / (2.0 * dy);
        float part3 = eps2_x[id] * p_x[id] + eps2_y[id] * p_y[id];
        float part4 = eps[id] * eps[id] * p_lap[id];

        float m = alpha / M_PI * atan(gamma * (T_eq - T[id]));

        float term1 = -part1 + part2 + part3 + part4;
        float term2 = p[id] * (1 - p[id]) * (p[id] - 0.5 + m);

        // Add noise using curand
        hiprandState localState = state[id];
        float noise = a * p[id] * (1 - p[id]) * (hiprand_uniform(&localState) - 0.5);
        
        dif_p[id] = dt / tau * (term1 + term2 + noise);

        p[id] = p[id] + dif_p[id];
    }
}

__global__ void T_field(float* T, float* d_eta, float* T_new, float hx, float hy, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        int left = idy * nx + ((idx - 1 + nx) % nx);
        int right = idy * nx + ((idx + 1) % nx);
        int top = ((idy - 1 + ny) % ny) * nx + idx;
        int bottom = ((idy + 1) % ny) * nx + idx;

        float lap = (T[top] + T[bottom] - 2.0 * T[id]) / (hx * hx) + 
                    (T[left] + T[right] - 2.0 * T[id]) / (hy * hy);

        T_new[id] = T[id] + dt * lap + K * d_eta[id];
    }
}

__global__ void zero_flux_BC(float* arr, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < nx && idy < ny) {
        int id = idy * nx + idx;

        if (idx == 0)
            arr[id] = arr[idy * nx + 1];
        else if (idx == nx - 1)
            arr[id] = arr[idy * nx + (nx - 2)];
        if (idy == 0)
            arr[id] = arr[1 * nx + idx];
        else if (idy == ny - 1)
            arr[id] = arr[(ny - 2) * nx + idx];
    }
}

__global__ void elementwise_multiply(float* aa, float* bb, float* result, int nx, int ny) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idy * nx + idx;

    if (idx < nx && idy < ny) {
        result[id] = aa[id] * bb[id];
    }
}

void printArray(const char* name, float* array, int nx, int ny) {
    std::cout << name << ":\n";
    for (int i = 0; i < ny; ++i) {
        for (int j = 0; j < nx; ++j) {
            std::cout << array[i * nx + j] << " ";
        }
        std::cout << "\n";
    }
}

void saveArray(const char* filename, float* array, int nx, int ny) {
    std::ofstream file(filename);
    for (int i = 0; i < ny; ++i) {
        for (int j = 0; j < nx; ++j) {
            file << array[i * nx + j] << " ";
        }
        file << "\n";
    }
    file.close();
}

int main() {
    float t = 0.0;

    // Initialize arrays
    float *T, *p, *theta, *p_x, *p_y, *p_lap, *eps, *eps_prime, *eps2_x, *eps2_y;
    float *d_T, *d_p, *d_theta, *d_p_x, *d_p_y, *d_p_lap, *d_dif_p, *d_eps, *d_eps_prime, *d_eps2, *d_eps2_x, *d_eps2_y;
    hiprandState* d_state;

    T = (float*)calloc(nx * ny, sizeof(float));
    p = (float*)calloc(nx * ny, sizeof(float));
    p_lap = (float*)calloc(nx * ny, sizeof(float));
    theta = (float*)calloc(nx * ny, sizeof(float));
    p_x = (float*)calloc(nx * ny, sizeof(float));
    p_y = (float*)calloc(nx * ny, sizeof(float));
    eps = (float*)calloc(nx * ny, sizeof(float));
    eps_prime = (float*)calloc(nx * ny, sizeof(float));
    eps2_x = (float*)calloc(nx * ny, sizeof(float));
    eps2_y = (float*)calloc(nx * ny, sizeof(float));

    // Define the center of the grid
    int centerX = nx / 2;
    int centerY = ny / 2;
    int radius = 5;

    // Loop over each element in the grid
    for (int i = 0; i < nx; ++i) {
        for (int j = 0; j < ny; ++j) {
            // Check if the current element is within the nucleation region
            if ((i - centerX) * (i - centerX) + (j - centerY) * (j - centerY) < radius * radius) {
                // Set the value to 1 for nucleation
                p[j * nx + i] = 1.0;
            }
        }
    }

    hipMalloc((void**)&d_T, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_lap, nx * ny * sizeof(float));
    hipMalloc((void**)&d_theta, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_x, nx * ny * sizeof(float));
    hipMalloc((void**)&d_p_y, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps_prime, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2_x, nx * ny * sizeof(float));
    hipMalloc((void**)&d_eps2_y, nx * ny * sizeof(float));
    hipMalloc((void**)&d_state, nx * ny * sizeof(hiprandState));

    hipMemcpy(d_T, T, nx * ny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, nx * ny * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    std::cout << "Number of blocks in x: " << numBlocks.x << std::endl;
    std::cout << "Number of blocks in y: " << numBlocks.y << std::endl;

    init_curand<<<numBlocks, threadsPerBlock>>>(d_state, time(0), nx, ny);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();

    for (int i = 0; i < nIter; ++i) {
        grad<<<numBlocks, threadsPerBlock>>>(d_p, d_p_x, d_p_y, hx, hy, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        laplace<<<numBlocks, threadsPerBlock>>>(d_p, d_p_lap, hx, hy, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        get_theta<<<numBlocks, threadsPerBlock>>>(d_p_x, d_p_y, d_theta, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        get_eps<<<numBlocks, threadsPerBlock>>>(d_theta, d_eps, d_eps_prime, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        elementwise_multiply<<<numBlocks, threadsPerBlock>>>(d_eps, d_eps, d_eps2, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        grad<<<numBlocks, threadsPerBlock>>>(d_eps2, d_eps2_x, d_eps2_y, hx, hy, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        std::cout << "before phase_field\n";

        phase_field<<<numBlocks, threadsPerBlock>>>(d_state, d_eps, d_eps_prime, d_eps2_x, d_eps2_y, d_p, d_p_x, d_p_y, d_p_lap, d_dif_p, d_T, hx, hy, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();


        std::cout << "after phase_field\n";

        zero_flux_BC<<<numBlocks, threadsPerBlock>>>(d_p, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        std::cout << "after zero flux bx p\n";

        T_field<<<numBlocks, threadsPerBlock>>>(d_T, d_p, d_T, hx, hy, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();

        std::cout << "after t_field update\n";

        zero_flux_BC<<<numBlocks, threadsPerBlock>>>(d_T, nx, ny);
        CUDA_CHECK_ERROR();
        hipDeviceSynchronize();
        std::cout << "after zero flux bx T\n";

        if (i % 200 == 0) {
            std::cout << "i: " << i << "\n";

            hipMemcpy(p_lap, d_p_lap, nx * ny * sizeof(float), hipMemcpyDeviceToHost);

            char filename[100];
            sprintf(filename, "ice/output_%d.txt", i);
            saveArray(filename, p_lap, nx, ny);
        }

        t += dt;
    }

    hipFree(d_T);
    hipFree(d_p);
    hipFree(d_theta);
    hipFree(d_p_x);
    hipFree(d_p_y);
    hipFree(d_eps);
    hipFree(d_eps_prime);
    hipFree(d_eps2_x);
    hipFree(d_eps2_y);
    hipFree(d_state);

    free(T);
    free(p);
    free(theta);
    free(p_x);
    free(p_y);
    free(eps);
    free(eps_prime);
    free(eps2_x);
    free(eps2_y);

    std::cout << "t=" << t << std::endl;
    return 0;
}